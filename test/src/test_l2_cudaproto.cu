/**
 * @file test_l2_cudaproto.cu
 * @author Jiannan Tian
 * @brief
 * @version 0.4
 * @date 2023-02-25
 *
 * (C) 2023 by Indiana University, Argonne National Laboratory
 *
 */

#include <cuda/std/functional>
#include <iostream>
#include <nvfunctional>
#include <string>
#include <typeinfo>

#include "common/capsule.hh"
#include "kernel/detail/lorenzo_proto.inl"

using std::cout;
using std::endl;
namespace proto = psz::cuda::__kernel::prototype;

using T  = float;
using FP = float;
using EQ = int32_t;

size_t t1d_len = 256;
dim3   t1d_len3{256, 1, 1};
dim3   t1d_stride3{1, 1, 1};
dim3   t1d_grid_dim{1, 1, 1};
dim3   t1d_block_dim{256, 1, 1};

size_t t2d_len = 256;
dim3   t2d_len3{16, 16, 1};
dim3   t2d_stride3{1, 16, 1};
dim3   t2d_grid_dim{1, 1, 1};
dim3   t2d_block_dim{16, 16, 1};

size_t t3d_len = 512;
dim3   t3d_len3{8, 8, 8};
dim3   t3d_stride3{1, 8, 64};
dim3   t3d_grid_dim{1, 1, 1};
dim3   t3d_block_dim{8, 8, 8};

#include "misc/correctness.inl"

bool test1(
    int         dim,
    T const*    h_input,
    size_t      len,
    dim3        len3,
    dim3        stride3,
    T const*    h_expected_output,
    std::string funcname)
{
    Capsule<T>  input(len);
    Capsule<T>  outlier(len);
    Capsule<EQ> eq(len);

    input.malloc().set_hptr(const_cast<T*>(h_input)).h2d();
    eq.malloc().mallochost();
    outlier.malloc();

    auto radius = 512;

    if (dim == 1)
        proto::c_lorenzo_1d1l<T>
            <<<t1d_grid_dim, t1d_block_dim>>>(input.dptr(), len3, stride3, radius, 1, eq.dptr(), outlier.dptr());
    else if (dim == 2)
        proto::c_lorenzo_2d1l<T>
            <<<t2d_grid_dim, t2d_block_dim>>>(input.dptr(), len3, stride3, radius, 1, eq.dptr(), outlier.dptr());
    else if (dim == 3)
        proto::c_lorenzo_3d1l<T>
            <<<t3d_grid_dim, t3d_block_dim>>>(input.dptr(), len3, stride3, radius, 1, eq.dptr(), outlier.dptr());
    hipDeviceSynchronize();

    eq.d2h();

    // for (auto i = 0; i < len; i++) { cout << h_eq[i] << endl; }

    bool ok = true;
    for (auto i = 0; i < len; i++) {
        // subject to change according to the algorithm
        if (eq.hptr(i) - radius != h_expected_output[i]) {
            ok = false;
            break;
        }
    }
    cout << funcname << " works as expected: " << (ok ? "yes" : "NO") << endl;

    input.free();
    eq.free().freehost();
    outlier.free();

    return ok;
}

bool test2(
    int         dim,
    EQ const*   _h_input,
    size_t      len,
    dim3        len3,
    dim3        stride3,
    T const*    h_expected_output,
    std::string funcname)
{
    auto radius = 512;

    Capsule<EQ> input(len);
    input.malloc().mallochost();

    for (auto i = 0; i < len; i++) input.hptr(i) = _h_input[i] + radius;
    input.h2d();

    Capsule<T> xdata(len);
    xdata.malloc().mallochost();

    if (dim == 1)
        proto::x_lorenzo_1d1l<T><<<t1d_grid_dim, t1d_block_dim>>>(
            input.dptr(), xdata.dptr() /* outlier */, len3, stride3, radius, 1, xdata.dptr());
    else if (dim == 2)
        proto::x_lorenzo_2d1l<T><<<t2d_grid_dim, t2d_block_dim>>>(
            input.dptr(), xdata.dptr() /* outlier */, len3, stride3, radius, 1, xdata.dptr());
    else if (dim == 3)
        proto::x_lorenzo_3d1l<T><<<t3d_grid_dim, t3d_block_dim>>>(
            input.dptr(), xdata.dptr() /* outlier */, len3, stride3, radius, 1, xdata.dptr());
    else {
        throw std::runtime_error("must be 1, 2, or 3D.");
    }

    hipDeviceSynchronize();

    xdata.d2h();

    bool ok = true;
    for (auto i = 0; i < len; i++) {
        if (xdata.hptr(i) != h_expected_output[i]) {
            ok = false;
            break;
        }
    }
    cout << funcname << " works as expected: " << (ok ? "yes" : "NO") << endl;

    input.free().freehost();
    xdata.free().freehost();

    return ok;
}

bool test3(int dim, T const* h_input, size_t len, dim3 len3, dim3 stride3, std::string funcname)
{
    Capsule<T> input(len);
    input.malloc().mallochost();
    for (auto i = 0; i < len; i++) input.hptr(i) = h_input[i];
    input.h2d();

    Capsule<EQ> eq(len);
    eq.malloc();

    Capsule<T> outlier(len);
    outlier.malloc();

    Capsule<T> xdata(len);
    xdata.malloc().mallochost();

    auto radius = 512;

    auto eb     = 1e-2;
    auto ebx2   = eb * 2;
    auto ebx2_r = 1 / (eb * 2);

    if (dim == 1) {
        proto::c_lorenzo_1d1l<T>
            <<<t1d_grid_dim, t1d_block_dim>>>(input.dptr(), len3, stride3, radius, ebx2_r, eq.dptr(), outlier.dptr());
        hipDeviceSynchronize();
        proto::x_lorenzo_1d1l<T><<<t1d_grid_dim, t1d_block_dim>>>(
            eq.dptr(), xdata.dptr() /* outlier */, len3, stride3, radius, ebx2, xdata.dptr());
        hipDeviceSynchronize();
    }
    else if (dim == 2) {
        proto::c_lorenzo_2d1l<T>
            <<<t2d_grid_dim, t2d_block_dim>>>(input.dptr(), len3, stride3, radius, ebx2_r, eq.dptr(), outlier.dptr());
        hipDeviceSynchronize();
        proto::x_lorenzo_2d1l<T><<<t2d_grid_dim, t2d_block_dim>>>(
            eq.dptr(), xdata.dptr() /* outlier */, len3, stride3, radius, ebx2, xdata.dptr());
        hipDeviceSynchronize();
    }
    else if (dim == 3) {
        proto::c_lorenzo_3d1l<T>
            <<<t3d_grid_dim, t3d_block_dim>>>(input.dptr(), len3, stride3, radius, ebx2_r, eq.dptr(), outlier.dptr());
        hipDeviceSynchronize();
        proto::x_lorenzo_3d1l<T><<<t3d_grid_dim, t3d_block_dim>>>(
            eq.dptr(), xdata.dptr() /* outlier */, len3, stride3, radius, ebx2, xdata.dptr());
        hipDeviceSynchronize();
    }

    // hipMemcpy(h_xdata, xdata, sizeof(EQ) * len, hipMemcpyDeviceToHost);
    xdata.d2h();

    // for (auto i = 0; i < len; i++) { cout << h_xdata[i] << endl; }

    bool ok = true;
    for (auto i = 0; i < len; i++) {
        if (xdata.hptr(i) != h_input[i]) {
            ok = false;
            break;
        }
    }
    cout << funcname << " works as expected: " << (ok ? "yes" : "NO") << endl;

    // input.debug();

    input.free().freehost();
    eq.free();
    outlier.free();
    xdata.free().freehost();

    return ok;
}

bool run_test1()
{
    auto all_pass = true;

    all_pass = all_pass and test1(1, t1d_in, t1d_len, t1d_len3, t1d_stride3, t1d_comp_out, "standalone cl1d1l");
    all_pass = all_pass and test1(2, t2d_in, t2d_len, t2d_len3, t2d_stride3, t2d_comp_out, "standalone cl2d1l");
    all_pass = all_pass and test1(3, t3d_in, t3d_len, t3d_len3, t3d_stride3, t3d_comp_out, "standalone cl3d1l");

    return all_pass;
}

bool run_test2()
{
    auto all_pass = true;

    all_pass = all_pass and test2(1, t1d_eq, t1d_len, t1d_len3, t1d_stride3, t1d_decomp_out, "standalone xl1d1l");
    all_pass = all_pass and test2(2, t2d_eq, t2d_len, t2d_len3, t2d_stride3, t2d_decomp_out, "standalone xl2d1l");
    all_pass = all_pass and test2(3, t3d_eq, t3d_len, t3d_len3, t3d_stride3, t3d_decomp_out, "standalone xl3d1l");

    return all_pass;
}

bool run_test3()
{
    auto all_pass = true;

    all_pass = all_pass and test3(1, t1d_in, t1d_len, t1d_len3, t1d_stride3, "lorenzo_1d1l");
    all_pass = all_pass and test3(2, t2d_in, t2d_len, t2d_len3, t2d_stride3, "lorenzo_2d1l");
    all_pass = all_pass and test3(3, t3d_in, t3d_len, t3d_len3, t3d_stride3, "lorenzo_3d1l");

    return all_pass;
}

int main()
{
    auto all_pass = true;

    all_pass = all_pass and run_test1();
    all_pass = all_pass and run_test2();
    all_pass = all_pass and run_test3();

    if (all_pass)
        return 0;
    else
        return -1;
}
