/**
 * @file hf_bookg.cu
 * @author Jiannan Tian
 * @brief
 * @version 0.3
 * @date 2022-11-03
 *
 * (C) 2022 by Indiana University, Argonne National Laboratory
 *
 */

#include "detail/hf_bookg.inl"
#include "hf/hf_bookg.hh"

#define PAR_BOOK(T, H) \
    template void asz::hf_buildbook_g<T, H>(uint32_t*, int const, H*, uint8_t*, int const, float*, hipStream_t);

PAR_BOOK(uint8_t, uint32_t);
PAR_BOOK(uint16_t, uint32_t);
PAR_BOOK(uint32_t, uint32_t);
PAR_BOOK(float, uint32_t);

PAR_BOOK(uint8_t, uint64_t);
PAR_BOOK(uint16_t, uint64_t);
PAR_BOOK(uint32_t, uint64_t);
PAR_BOOK(float, uint64_t);

PAR_BOOK(uint8_t, unsigned long long);
PAR_BOOK(uint16_t, unsigned long long);
PAR_BOOK(uint32_t, unsigned long long);
PAR_BOOK(float, unsigned long long);

#undef PAR_BOOK
